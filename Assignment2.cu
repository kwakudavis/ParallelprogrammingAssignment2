#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>
#include <vector>
#include <string>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "jbutil.h"


using namespace std;




using namespace std;



//First Iteration function global fucntion

__global__ void firstIteration ( int*array, int rows, int columns,bool state){

       int blackPixCounter;



       for (int i = 0; i < rows; i++){
                  for (int j  = 0; j < columns; j++)  {



                    //If the pixel is black

                    if(array[i*columns+j] == 1){



                    //Black pixel counter
                       blackPixCounter=0;

                    ///Counting number of black neighbors



                    //If north is black, increase black neighbor pixel counter
                    if((i-1)>0){

                          if( array[(i-1)*columns+j] ==0){

                                  blackPixCounter++;
                            }

                    }

                    //If south is black,increase pixel neighbor counter
                    if((i+1)<rows){


                        if(array[(i+1)*columns+j] == 0){
                            blackPixCounter++;
                        }
                    }


                    //If east is black,increase pixel neighbor counter
                    if((j+1)<columns){


                         if (array[i*columns+(j+1)]==0){
                           blackPixCounter++;
                        }



                    }


                    //If west is black,increase pixel neighbor counter
                    if((j-1)>0){


                          if(array[i*columns+(j-1)]==0){

                             blackPixCounter++;
                          }

                      }


                    if( blackPixCounter > 2 && blackPixCounter <= 6){
                        state = true;

                    }



                    }


                  //Set element to 0 if condition holds
                        if(state){
                          array[i*columns+j] = 0;
                        }

                  state = false;

                  }


          }








}




//Second Iteration function global function

__global__ void secondIteration ( int*array, int rows, int columns,bool state){



         int blackPixCounter;



         for (int i = 0; i < rows; i++){
                    for (int j  = 0; j < columns; j++)  {



                      //If the pixel is black

                      if(array[i*columns+j] == 1){



                      //Black pixel counter
                         blackPixCounter=0;

                      ///Counting number of black neighbors



                      //If north is black, increase black neighbor pixel counter
                      if((i-1)>0){

                            if( array[(i-1)*columns+j] ==0){

                                    blackPixCounter++;
                              }

                      }

                      //If south is black,increase pixel neighbor counter
                      if((i+1)<rows){


                          if(array[(i+1)*columns+j] == 0){
                              blackPixCounter++;
                          }
                      }


                      //If east is black,increase pixel neighbor counter
                      if((j+1)<columns){


                           if (array[i*columns+(j+1)]==0){
                             blackPixCounter++;
                          }



                      }


                      //If west is black,increase pixel neighbor counter
                      if((j-1)>0){


                            if(array[i*columns+(j-1)]==0){

                               blackPixCounter++;
                            }

                        }


                      if( blackPixCounter > 2 && blackPixCounter <= 6){
                          state = true;

                      }



                      }


                    //Set element to 0 if condition holds
                          if(state){
                            array[i*columns+j] = 0;
                          }

                    state = false;

                    }


            }






}
















//Executes Zhang Suen Thinning Algorithm on each pixel
int ZhangSuenThinningAlgorithm( int*array, int rows, int columns){

    cout <<" The Output is "<<endl;
    bool state = true;



    //First iteration


   firstIteration(array,rows, columns, state);


    //Second iteration

  secondIteration(array,rows, columns, state);





    return 0;

}


//vector<vector<int> > &array
int printRestults(int*array, int rows, int columns,string filename){


       ofstream myfile;
       myfile.open (filename+"Results");




       //Displaying pixel values
        for (int i = 0; i < rows; i++){
                for (int j  = 0; j < columns; j++)  {
                //   cout<< array[i][j]<<" ";
              //     myfile<< array[i][j]<<" ";

              //  arr1[rowCounter*columns+columnCounter]

                cout<< array[i*columns+j]<<" ";
                myfile<< array[i*columns+j]<<" ";


                }
                cout <<endl;
                myfile << endl;

        }

        myfile.close();
        return 0;
}


bool isZeroOrOne(string word){
    if( ! (word.compare("0") || word.compare("1"))){
        return false;
    }

    return true;
}



bool isDigit(const string s){
    return !s.empty() && std::all_of(s.begin(), s.end(), ::isdigit);

}




//Main Program
int main(int argc, char *argv[]){


        //Request file name to be passed
        if(argc < 2){
            cout << "Enter a filename and reload" <<endl;
            return 0;
        }


        //Read filename
        string Filename = argv[1];

        //Create a file read stream
        ifstream File (Filename);




        //Return error message if file could not be opened

        if(!File.is_open()){
            cout<<"File not found"<<endl;
            return 0;
        }


        //Read first line, which should contain number of rows and columns.
        string Line;
        getline (File,Line);


        //Read first string in line
        istringstream iss(Line);


        //Get number of rows
        string word;
        iss >> word;

        //Initialization of row and column variables
        size_t rows = 0;
        size_t columns = 0;


        //check validity of word
        if(!isDigit(word)){
            cout << "Row value is not a number"<<endl;
            return 0;
        }
        rows = stoi(word);



        //Get number of columns
        iss >> word;
        if(!isDigit(word)){
            cout << "Column value is not a number"<<endl;
            return 0;
        }
        columns = stoi(word);

      //  cout<<"number of columns:";
    //    cout<<columns<<endl;

    //    cout<<"number of Rows:";
    //    cout<<rows<<endl;

        //Initialization of vector of vectors to hold pixel values
       vector <vector <int> > arr(rows, vector<int>(columns));



       //Allocation of host memory
        int *arr1;
        int *d_a;
        const int size = rows * columns * sizeof(int);
        arr1  = (int*)malloc(sizeof(int) * rows * columns);




        // Allocate device memory
        hipMalloc((void**)&d_a, sizeof(int) * rows * columns);



        /**

        // Transfer data from host to device memory
        hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

        // Executing kernel
        vector_add<<<1,1>>>(d_out, d_a, d_b, N);



        // Transfer data back to host memory
        hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);


            printf("out[0] = %f\n", out[0]);
            printf("PASSED\n");

            // Deallocate device memory
            hipFree(d_a);
            hipFree(d_b);
            hipFree(d_out);

            // Deallocate host memory
            free(a);
            free(b);
            free(out);

**/







        //Get remaining lines

        int rowCounter = 0;
        int columnCounter = 0;

        //while there are still more lines to read get words
        while(File.eof() != true){
            getline(File,Line);
            istringstream iss(Line);

                while(iss >> word){




                    //Return error message if a non digit is read

                  //  cout<<word;

                    if(!isZeroOrOne(word)){
                        cout<<"Invalid number detected in input"<<endl;
                        return 0;
                    }

                    //Convert to string and store value
                    arr[rowCounter][columnCounter] = stoi(word);

                    arr1[rowCounter*columns+columnCounter] = stoi(word);
                    columnCounter++;


                  }

            //       cout<<"columnCounter:"<<endl;
              //     cout<<columnCounter<<endl;

            //       cout<<"Moving to next Row"<<endl;
                    //Raise error if number of columns read does not match column number input
                    if(columnCounter != columns){
                        cout << " Column value does not match number of columns read"<<endl;
                       return 0;
                    }

                    //Reset Column counter
                    columnCounter = 0;


                    rowCounter++;


      }

           //Raise error if number of rows read does not match column number input
        if(rowCounter != rows ){
                cout << "Row value do noes not match number of rows read" <<endl;
                return 0;
        }







//Start Timer

double time = jbutil::gettime();
        //Applying thinning algorithm on each pixel value
       ZhangSuenThinningAlgorithm(arr1, rows, columns);


//Stop timer

time = jbutil::gettime() - time;
       //Display results and store in file
       printRestults(arr1,rows, columns,Filename);

       cout<<"Time taken is:"<<time<<"s"<<endl;





       return 1;

}
