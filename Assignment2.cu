#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>
#include <vector>
#include <string>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "jbutil.h"


using namespace std;




using namespace std;



//First Iteration function global fucntion

__global__ void firstIteration ( int*array, int rows, int columns,bool state){

       int blackPixCounter;



       for (int i = 0; i < rows; i++){
                  for (int j  = 0; j < columns; j++)  {



                    //If the pixel is black

                    if(array[i*columns+j] == 1){



                    //Black pixel counter
                       blackPixCounter=0;

                    ///Counting number of black neighbors



                    //If north is black, increase black neighbor pixel counter
                    if((i-1)>0){

                          if( array[(i-1)*columns+j] ==0){

                                  blackPixCounter++;
                            }

                    }

                    //If south is black,increase pixel neighbor counter
                    if((i+1)<rows){


                        if(array[(i+1)*columns+j] == 0){
                            blackPixCounter++;
                        }
                    }


                    //If east is black,increase pixel neighbor counter
                    if((j+1)<columns){


                         if (array[i*columns+(j+1)]==0){
                           blackPixCounter++;
                        }



                    }


                    //If west is black,increase pixel neighbor counter
                    if((j-1)>0){


                          if(array[i*columns+(j-1)]==0){

                             blackPixCounter++;
                          }

                      }


                    if( blackPixCounter > 2 && blackPixCounter <= 6){
                        state = true;

                    }



                    }


                  //Set element to 0 if condition holds
                        if(state){
                          array[i*columns+j] = 0;
                        }

                  state = false;

                  }


          }








}




//Second Iteration function global function

__global__ void secondIteration ( int*array, int rows, int columns,bool state){



         int blackPixCounter;



         for (int i = 0; i < rows; i++){
                    for (int j  = 0; j < columns; j++)  {



                      //If the pixel is black

                      if(array[i*columns+j] == 1){



                      //Black pixel counter
                         blackPixCounter=0;

                      ///Counting number of black neighbors



                      //If north is black, increase black neighbor pixel counter
                      if((i-1)>0){

                            if( array[(i-1)*columns+j] ==0){

                                    blackPixCounter++;
                              }

                      }

                      //If south is black,increase pixel neighbor counter
                      if((i+1)<rows){


                          if(array[(i+1)*columns+j] == 0){
                              blackPixCounter++;
                          }
                      }


                      //If east is black,increase pixel neighbor counter
                      if((j+1)<columns){


                           if (array[i*columns+(j+1)]==0){
                             blackPixCounter++;
                          }



                      }


                      //If west is black,increase pixel neighbor counter
                      if((j-1)>0){


                            if(array[i*columns+(j-1)]==0){

                               blackPixCounter++;
                            }

                        }


                      if( blackPixCounter >= 2 && blackPixCounter <= 6){
                          state = true;

                      }



                      }


                    //Set element to 0 if condition holds
                          if(state){
                            array[i*columns+j] = 0;
                          }

                    state = true;

                    }


            }






}
















//Executes Zhang Suen Thinning Algorithm on each pixel
int ZhangSuenThinningAlgorithm( int*array, int rows, int columns){

    cout <<" The Output is "<<endl;
    bool state = true;



    //First iteration


  // firstIteration(array,rows, columns, state);


   // Executing kernel
  firstIteration<<<1,1>>>(array, rows, columns, state);


    //Second iteration



    // Executing kernel
   secondIteration<<<1,1>>>(array, rows, columns, state);


//  secondIteration(array,rows, columns, state);





    return 0;

}


//vector<vector<int> > &array
int printRestults(int*array, int rows, int columns,string filename){


       ofstream myfile;
       myfile.open (filename+"Results");




       //Displaying pixel values
        for (int i = 0; i < rows; i++){
                for (int j  = 0; j < columns; j++)  {
                //   cout<< array[i][j]<<" ";
              //     myfile<< array[i][j]<<" ";

              //  arr1[rowCounter*columns+columnCounter]

                cout<< array[i*columns+j]<<" ";
                myfile<< array[i*columns+j]<<" ";


                }
                cout <<endl;
                myfile << endl;

        }

        myfile.close();
        return 0;
}


bool isZeroOrOne(string word){
    if( ! (word.compare("0") || word.compare("1"))){
        return false;
    }

    return true;
}



bool isDigit(const string s){
    return !s.empty() && std::all_of(s.begin(), s.end(), ::isdigit);

}




//Main Program
int main(int argc, char *argv[]){


        //Request file name to be passed
        if(argc < 2){
            cout << "Enter a filename and reload" <<endl;
            return 0;
        }


        //Read filename
        string Filename = argv[1];

        //Create a file read stream
        ifstream File (Filename);




        //Return error message if file could not be opened

        if(!File.is_open()){
            cout<<"File not found"<<endl;
            return 0;
        }


        //Read first line, which should contain number of rows and columns.
        string Line;
        getline (File,Line);


        //Read first string in line
        istringstream iss(Line);


        //Get number of rows
        string word;
        iss >> word;

        //Initialization of row and column variables
        size_t rows = 0;
        size_t columns = 0;


        //check validity of word
        if(!isDigit(word)){
            cout << "Row value is not a number"<<endl;
            return 0;
        }
        rows = stoi(word);



        //Get number of columns
        iss >> word;
        if(!isDigit(word)){
            cout << "Column value is not a number"<<endl;
            return 0;
        }
        columns = stoi(word);

      //  cout<<"number of columns:";
    //    cout<<columns<<endl;

    //    cout<<"number of Rows:";
    //    cout<<rows<<endl;

        //Initialization of vector of vectors to hold pixel values
       vector <vector <int> > arr(rows, vector<int>(columns));



       //Allocation of host memory
        int *arr1;

    //    int *d_rows;
    //    int *d_columns;


        //const int size = rows * columns * sizeof(int);
        arr1  = (int*)malloc(sizeof(int) * rows * columns);
      //  d_rows = (int*)malloc(sizeof(int));
      //  d_columns = (int*)malloc(sizeof(int));




        // Allocate device memory
        int *d_a;
        hipMalloc((void**)&d_a, sizeof(int) * rows * columns);
    //    hipMalloc((void**)&d_rows, sizeof(int));
    //    hipMalloc((void**)&d_columns, sizeof(int));















        //Get remaining lines



          int columnCounter = 0;
          int rowCounter = 0;

        //while there are still more lines to read get words
        while(File.eof() != true){
            getline(File,Line);
            istringstream iss(Line);

                while(iss >> word){




                    //Return error message if a non digit is read

                  //  cout<<word;

                    if(!isZeroOrOne(word)){
                        cout<<"Invalid number detected in input"<<endl;
                        return 0;
                    }

                    //Convert to string and store value
                    arr[rowCounter][columnCounter] = stoi(word);

                    arr1[rowCounter*columns+columnCounter] = stoi(word);
                    columnCounter++;


                  }

            //       cout<<"columnCounter:"<<endl;
              //     cout<<columnCounter<<endl;

            //       cout<<"Moving to next Row"<<endl;
                    //Raise error if number of columns read does not match column number input
                    if(columnCounter != columns){
                        cout << " Column value does not match number of columns read"<<endl;
                       return 0;
                    }

                    //Reset Column counter
                    columnCounter = 0;


                    rowCounter++;


      }

           //Raise error if number of rows read does not match column number input
        if(rowCounter != rows ){
                cout << "Row value do noes not match number of rows read" <<endl;
                return 0;
        }




      // Transfer data from host to device memory
      hipMemcpy(d_a, arr1, sizeof(int) * rows * columns, hipMemcpyHostToDevice);


//Start Timer

double time = jbutil::gettime();

        //Applying thinning algorithm on each pixel value
       ZhangSuenThinningAlgorithm(arr1, rows, columns);






       // Transfer data back to host memory
       hipMemcpy(arr1, d_a, sizeof(int), hipMemcpyDeviceToHost);



           printf("PASSED\n");








//Stop timer

time = jbutil::gettime() - time;
       //Display results and store in file
       printRestults(arr1,rows, columns,Filename);

       cout<<"Time taken is:"<<time<<"s"<<endl;



       // Deallocate device memory
       hipFree(d_a);

       // Deallocate host memory
       free(arr1);



       return 1;

}
